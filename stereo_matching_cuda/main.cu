#include "rgb_to_grayscale.cuh"
#include "stb_image.h"
#include "stb_image_write.h"
#include "filter.cuh"
#include "costVolume.cuh"
#include "systemIncludes.h"
#include "guidedFilter.cuh"
#include "occlusion.cuh"

// int stbi_write_png(char const *filename, int w, int h, int comp, const void *data, int stride_in_bytes);
// DOCUMENTATION
//
// Limitations:
//    - no 12-bit-per-channel JPEG
//    - no JPEGs with arithmetic coding
//    - GIF always returns *comp=4
//
// Basic usage (see HDR discussion below for HDR usage):
//    int x,y,n;
//    unsigned char *data = stbi_load(filename, &x, &y, &n, 0);
//    // ... process data if not NULL ...
//    // ... x = width, y = height, n = # 8-bit components per pixel ...
//    // ... replace '0' with '1'..'4' to force that many components per pixel
//    // ... but 'n' will always be the number that it would have been if you said 0
//    stbi_image_free(data)
//
// Standard parameters:
//    int *x                 -- outputs image width in pixels
//    int *y                 -- outputs image height in pixels
//    int *channels_in_file  -- outputs # of image components in image file
//    int desired_channels   -- if non-zero, # of image components requested in result
//
// The return value from an image loader is an 'unsigned char *' which points
// to the pixel data, or NULL on an allocation failure or if the image is
// corrupt or invalid. The pixel data consists of *y scanlines of *x pixels,
// with each pixel consisting of N interleaved 8-bit components; the first
// pixel pointed to is top-left-most in the image. There is no padding between
// image scanlines or between pixels, regardless of format. The number of
// components N is 'desired_channels' if desired_channels is non-zero, or
// *channels_in_file otherwise. If desired_channels is non-zero,
// *channels_in_file has the number of components that _would_ have been
// output otherwise. E.g. if you set desired_channels to 4, you will always
// get RGBA output, but you can check *channels_in_file to see if it's trivially
// opaque because e.g. there were only 3 channels in the source image.
//
// An output image with N components has the following components interleaved
// in this order in each pixel:
//
//     N=#comp     components
//       1           grey
//       2           grey, alpha
//       3           red, green, blue
//       4           red, green, blue, alpha
//
// If image loading fails for any reason, the return value will be NULL,
// and *x, *y, *channels_in_file will be unchanged. The function
// stbi_failure_reason() can be queried for an extremely brief, end-user
// unfriendly explanation of why the load failed. Define STBI_NO_FAILURE_STRINGS
// to avoid compiling these strings at all, and STBI_FAILURE_USERMSG to get slightly
// more user-friendly ones.
//
// Paletted PNG, BMP, GIF, and PIC images are automatically depalettized.

int main(int argc, char **argv)
{
	bool host_compare = true;
	int wRadius = 3;

	//// Image loading
	int width;
	int height;
	int channels_in_file;

	unsigned char *data = stbi_load("im2.png", &width, &height, &channels_in_file, 0);

	//// Split the image in several channels

	// Make an array for each channel
	unsigned char **splitted_data = new unsigned char *[channels_in_file];
	for (size_t channel = 0; channel < channels_in_file; ++channel)
	{
		splitted_data[channel] = new unsigned char[width*height];
	}

	// Fill the arrays
	for (size_t row = 0; row < height; ++row)
	{
		for (size_t col = 0; col < width; ++col)
		{
			for (size_t channel = 0; channel < channels_in_file; ++channel)
			{
				splitted_data[channel][row * width + col] = data[channel + (row * (width * channels_in_file) + col * channels_in_file)];
			}
		}
	}

	printf("Starting...\n");

	// set up devices
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));
	//end setup device

	//Warmup
	int n = width * height;
	cout << "Warmup ..." << endl;
	unsigned char* grayscale = rgb_to_grayscale(data, n, channels_in_file, !host_compare);
	stbi_write_png("./data/warmup.png", width, height, 1, grayscale, 0);
	//end warmup

	/**
	///////////////////////////////////////////////////////////////////////////
	// Write each channel in separated files
	for (size_t channel = 0; channel < channels_in_file; ++channel)
	{
		std::string filename("image_" + std::to_string(channel) + ".png");
		stbi_write_png(filename.c_str(), width, height, 1, splitted_data[channel], 0);
	}

	//// Reconstruct the original image from splitted images
	unsigned char *copy_data = new unsigned char[channels_in_file * width * height];
	for (size_t row = 0; row < height; ++row)
	{
		for (size_t col = 0; col < width; ++col)
		{
			for (size_t channel = 0; channel < channels_in_file; ++channel)
			{
				copy_data[channel + (row * (width * channels_in_file) + col * channels_in_file)] = splitted_data[channel][(row * width + col)];
			}
		}
	}

	stbi_write_png("image_copy.png", width, height, channels_in_file, copy_data, 0);

	//// Free the memory
	for (int channel = 0; channel < channels_in_file; ++channel)
	{
		delete[] splitted_data[channel];
	}
	delete[] splitted_data;

	delete copy_data;
	////////////////////////////////////////////////////////////////////////////////////////////////
	**/

	//Begin
	// Image loading
	int w1, h1, ch1;
	int w2, h2, ch2;
	unsigned char *data1 = stbi_load("./data/tsukuba0.png", &w1, &h1, &ch1, 0);
	unsigned char *data2 = stbi_load("./data/tsukuba1.png", &w2, &h2, &ch2, 0);
	int n1 = w1 * h1;
	int n2 = w2 * h2;

	cout << "Resolution : " << w1 << "x" << h1 << endl;
	//rgb to grayscale
	cout << "RGB to grayscale ..." << endl;
	unsigned char* I_l = rgb_to_grayscale(data1, n1, ch1, host_compare);
	unsigned char* I_r = rgb_to_grayscale(data2, n2, ch2, host_compare);
	//end rgb to grayscale

	//Cost volume
	int size_d = D_MAX - D_MIN + 1;
	int totalSize1 = n1 * size_d;
	int totalSize2 = n2 * size_d;
	float* costl = (float*)malloc(sizeof(float)*totalSize1);
	float* costr = (float*)malloc(sizeof(float)*totalSize2);
	memset(costl, 0, sizeof(float)*totalSize1);
	memset(costr, 0, sizeof(float)*totalSize2);
	cout << "Cost Volume ..." << endl;

	const int dminl = D_MIN;
	compute_cost(I_l, I_r, costl, w1, w2, h1, h2, dminl, host_compare);
	const int dminr = -D_MAX;
	compute_cost(I_r, I_l, costr, w2, w1, h2, h1, dminr, host_compare);
	//end cost volume

	//guided Filter
	unsigned char* mean1 = (unsigned char*)malloc(n1);
	unsigned char* mean2 = (unsigned char*)malloc(n2);
	float* filtered_costl = (float*)malloc(sizeof(float) * totalSize1);
	float* filtered_costr = (float*)malloc(sizeof(float) * totalSize2);
	memset(mean1, 0, sizeof(unsigned char)*n1);
	memset(mean2, 0, sizeof(unsigned char)*n2);
	memset(filtered_costl, 0, sizeof(float)*totalSize1);
	memset(filtered_costr, 0, sizeof(float)*totalSize2);
	cout << "guided filter ..." << endl;
	compute_guided_filter(I_l, costl, filtered_costl, mean1, (const int)w1, (const int)h1, (const int)size_d, host_compare);
	compute_guided_filter(I_r, costr, filtered_costr, mean2, (const int)w2, (const int)h2, (const int)size_d, host_compare);

	//unsigned char* mean = (unsigned char*)malloc(height*width); //osef
	//memset(mean, 0, sizeof(unsigned char)*totalSize1);
	//compute_guided_filter(grayscale, cost, filtered, mean, (const int)width, (const int)height, (const int)size_d, host_compare);
	//stbi_write_png("./data/uhd_mean.png", width, height, 1, mean, 0);
	//free(mean);


	//end guided Filter

	float* best_costl = (float*)malloc(n1 * sizeof(float));
	float* best_costr = (float*)malloc(n2 * sizeof(float));
	memset(best_costl, 9999999.0f, n1 * sizeof(float));
	memset(best_costr, 9999999.0f, n2 * sizeof(float));

	float* dmapl = (float*)malloc(n1 * sizeof(float));
	float* dmapr = (float*)malloc(n2 * sizeof(float));
	unsigned char* dmaplChar = (unsigned char*)malloc(n1);
	unsigned char* dmaprChar = (unsigned char*)malloc(n2);
	memset(dmapl, 0, n1 * sizeof(float));
	memset(dmapr, 0, n2 * sizeof(float));
	memset(dmaplChar, 0, n1);
	memset(dmaprChar, 0, n2);
	disparity_selection(filtered_costl, best_costl, dmapl, (const int)w1, (const int)h1, dminl, host_compare);
	disparity_selection(filtered_costr, best_costr, dmapr, (const int)w2, (const int)h2, dminr, host_compare);
	//for (int i = 0; i < n1; i++) { cout << best_costl[i] << endl; }

	//const int dOcclusion = 2 * size_d;
	const int dOcclusion = (dminl - 1);
	detect_occlusion(dmapl, dmapr, dOcclusion, dmaplChar, dmaprChar, w1, h1);
	int vMin = D_MIN;
	fill_occlusion(dmapl, w1, h1, vMin);

	//write images
	cout << "writing images ..." << endl;
	stbi_write_png("./data/image_left.png", w1, h1, 1, I_l, 0);
	stbi_write_png("./data/image_right.png", w2, h2, 1, I_r, 0);
	stbi_write_png("./data/image_mean_left.png", w1, h1, 1, mean1, 0);
	stbi_write_png("./data/image_mean_right.png", w2, h2, 1, mean2, 0);
	stbi_write_png("./data/disparity_map_left.png", w1, h1, 1, dmaplChar, 0);
	stbi_write_png("./data/disparity_map_right.png", w2, h2, 1, dmaprChar, 0);
	//end writing images

	//free the memory
	cout << "Free the memory ..." << endl;
	free(grayscale);
	stbi_image_free(data);
	free(I_l);
	free(I_r);
	stbi_image_free(data1);
	stbi_image_free(data2);
	free(mean1);
	free(mean2);
	free(costl);
	free(costr);
	free(filtered_costl);
	free(filtered_costr);
	free(dmapl);
	free(dmapr);
	free(best_costr);
	free(best_costl);
	free(dmaprChar);
	free(dmaplChar);

	return 0;
}