#include "hip/hip_runtime.h"
#include "guidedFilter.cuh"

void compute_guided_filter(unsigned char* i, float* cost, float* filter_cost, float* disp_map, unsigned char* mean, const int w, const int h, const int size_d, int dmin, bool host_gpu_compare) {
	int n = w * h;
	int volume = size_d * w*h;
	int n_fl = sizeof(float)*n;
	int radius = 1 * RADIUS;
	//1st step : compute mean filter and its covariance
	// GPU var
	unsigned char* d_i;
	unsigned char* d_mean;
	float* d_im;
	float* d_mean_im;
	float* d_var_im;
	float* d_cost;
	float* d_filt_cost;
	float* d_dmap;

	//CPU var
	float* fMean = (float*)malloc(n_fl);

	float* h_im = (float*)malloc(n_fl);
	float* h_mean_im = (float*)malloc(n_fl);
	float* h_var_im = (float*)malloc(n_fl);
	float* empty = (float*)malloc(n_fl);
	float* big_empty = (float*)malloc(size_d*n_fl);

	//memset
	memset(fMean, 0, n);
	memset(mean, 0, n);
	memset(h_im, 0.0f, n_fl);
	memset(h_mean_im, 0.0f, n_fl);
	memset(h_var_im, 0.0f, n_fl);
	memset(empty, 0.0f, n_fl);
	memset(big_empty, 0.0f, n_fl*size_d);

	//cuda malloc
	CHECK(hipMalloc((unsigned char**)&d_i, n));
	CHECK(hipMalloc((unsigned char**)&d_mean, n));
	CHECK(hipMalloc((void**)&d_im, n_fl));
	CHECK(hipMalloc((void**)&d_mean_im, n_fl));
	CHECK(hipMalloc((void**)&d_var_im, n_fl));
	CHECK(hipMalloc((void**)&d_cost, size_d*n_fl));
	CHECK(hipMalloc((void**)&d_filt_cost, n_fl));
	CHECK(hipMalloc((void**)&d_dmap, n_fl));

	//cuda memcpy host -> device
	CHECK(hipMemcpy(d_i, i, n, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_mean, mean, n, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_im, h_im, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_mean_im, h_mean_im, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_filt_cost, filter_cost, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_dmap, disp_map, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_cost, cost, size_d*n_fl, hipMemcpyHostToDevice));

	float* h_filt_cost = (float*)malloc(n_fl * sizeof(float));
	memset(h_filt_cost, 0, sizeof(float)*(n_fl));

	CHECK(hipMemcpy(h_filt_cost, d_filt_cost, n_fl, hipMemcpyDeviceToHost));

	dim3 blockDim(128);

	dim3 gridDim((n + blockDim.x - 1) / blockDim.x);
	//im unsigned char -> float
	chToFlOnGPU << <gridDim, blockDim >> > (d_i, d_im, n);

	//Compute Integral im1
	CHECK(hipMemcpy(h_im, d_im, n_fl, hipMemcpyDeviceToHost));

	float* integral_im = (float*)malloc(n_fl);
	memset(integral_im, 0.0f, n_fl);
	integral(h_im, integral_im, w, h);

	float* integral_imCPU = (float*)malloc(n_fl);
	memset(integral_imCPU, 0.0f, n_fl);
	integralOnCPU(h_im, integral_imCPU, w, h);
	check_errors(integral_im, integral_imCPU, w * h);


	float* d_integral_im;
	CHECK(hipMalloc((void**)&d_integral_im, n_fl));
	CHECK(hipMemcpy(d_integral_im, integral_im, n_fl, hipMemcpyHostToDevice));
	dim3 y(16, 16);
	dim3 x((w + y.x - 1) / y.x, (h + y.y - 1) / y.y);
	computeBoxFilterOnGPU << < x, y >> > (d_im, d_integral_im, d_mean_im, (const int)w, (const int)h);

	CHECK(hipMemcpy(fMean, d_mean_im, n, hipMemcpyDeviceToHost));
	

	computeBoxFilterOnCPU(h_im, integral_imCPU, h_mean_im, (const int)w, (const int)h);
	//for (size_t i = 0; i < n; i++)
	//{
	//	cout << fMean[i] << endl;
	//}
	bool verif = check_errors(h_mean_im, fMean, n);
	if (verif) cout << "Box filter ok!" << endl;


	gridDim.x = (n + blockDim.x - 1) / blockDim.x;
	flToChOnGPU << <gridDim, blockDim >> > (d_mean_im, d_mean, n);

	//compute variance
	float* d_imSquare;
	float* d_meanSquare;
	float* d_integral_square;
	float* d_temp;
	float* imSquare = (float*)malloc(n_fl);
	float* integral_square = (float*)malloc(n_fl);
	memset(integral_square, 0, n_fl);
	memset(imSquare, 0, n_fl);
	CHECK(hipMalloc((void**)&d_imSquare, n_fl));
	CHECK(hipMalloc((void**)&d_meanSquare, n_fl));
	CHECK(hipMalloc((void**)&d_integral_square, n_fl));
	CHECK(hipMalloc((void**)&d_temp, n_fl));
	CHECK(hipMemcpy(d_imSquare, empty, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_meanSquare, empty, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_integral_square, empty, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_temp, empty, n_fl, hipMemcpyHostToDevice));
	dim3 mult(1024);
	dim3 gmult((n + mult.x - 1) / mult.x);

	// I*I AND mean*mean
	pixelMultOnGPU << < gmult, mult >> > (d_im, d_im, d_imSquare, n);
	pixelMultOnGPU << < gmult, mult >> > (d_mean_im, d_mean_im, d_meanSquare, n);
	CHECK(hipMemcpy(imSquare, d_imSquare, n_fl, hipMemcpyDeviceToHost));

	//mean(I*I)
	integral(imSquare, integral_square, w, h);
	CHECK(hipMemcpy(d_integral_square, integral_square, n_fl, hipMemcpyHostToDevice));
	computeBoxFilterOnGPU << < x, y >> > (d_imSquare, d_integral_square, d_temp, (const int)w, (const int)h);

	//var = mean(I*I) - mean*mean
	pixelSousOnGPU << <gridDim, blockDim >> > (d_temp, d_meanSquare, d_var_im, n);
	CHECK(hipMemcpy(mean, d_mean, n, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_var_im, d_var_im, n_fl, hipMemcpyDeviceToHost));

	//compute pk, a_k and b_k, a_i, b_i, q

	//1. variable in device
	float* d_ak;
	float* d_bk;
	float* d_ak_int;
	float* d_bk_int;
	float* d_ak_mean;
	float* d_bk_mean;
	float* d_pk;
	float* d_pki;
	float* d_pki_mean;
	float* d_convol;
	float* d_convol_int;
	float* d_q;
	float* d_convol_mean;
	float* d_pki_int;
	//variable on cpu
	float* pki_int = (float*)malloc(n_fl);
	float* pki = (float*)malloc(n_fl);
	float* convol = (float*)malloc(n_fl);
	float* convol_int = (float*)malloc(n_fl);
	float* ak = (float*)malloc(n_fl);
	float* bk = (float*)malloc(n_fl);
	float* ak_int = (float*)malloc(n_fl);
	float* bk_int = (float*)malloc(n_fl);

	CHECK(hipMalloc((void**)&d_pki, n_fl*size_d));
	CHECK(hipMalloc((void**)&d_pki_int, n_fl));
	CHECK(hipMalloc((void**)&d_pki_mean, n_fl));
	CHECK(hipMalloc((void**)&d_ak, n_fl));
	CHECK(hipMalloc((void**)&d_bk, n_fl));
	CHECK(hipMalloc((void**)&d_ak_int, n_fl));
	CHECK(hipMalloc((void**)&d_bk_int, n_fl));
	CHECK(hipMalloc((void**)&d_ak_mean, n_fl));
	CHECK(hipMalloc((void**)&d_bk_mean, n_fl));
	CHECK(hipMalloc((void**)&d_convol, n_fl));
	CHECK(hipMalloc((void**)&d_convol_int, n_fl));
	CHECK(hipMalloc((void**)&d_convol_mean, n_fl));
	CHECK(hipMalloc((void**)&d_q, n_fl));
	dim3 bdim(1024);
	dim3 gdim((n + bdim.x - 1) / bdim.x);
	dim3 bdim2(16, 16);
	dim3 gdim2((w + bdim2.x - 1) / bdim2.x, (h + bdim2.y - 1) / bdim2.y);

	//loop over d range
	for (int s = 0; s < size_d; s++) {
		int start = s * n;
		memset(pki_int, 0.0f, n_fl);
		memset(pki, 0.0f, n_fl);
		memset(convol, 0.0f, n_fl);
		memset(convol_int, 0.0f, n_fl);
		memset(ak, 0.0f, n_fl);
		memset(ak_int, 0.0f, n_fl);
		memset(bk, 0.0f, n_fl);
		memset(bk_int, 0.0f, n_fl);
		CHECK(hipMemcpy(d_pki, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pki_int, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_pki_mean, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_convol, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_convol_int, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_convol_mean, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_ak, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_bk, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_ak_int, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_bk_int, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_ak_mean, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_bk_mean, empty, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_q, empty, n_fl, hipMemcpyHostToDevice));

		//Cost -> pk
		copyFromBigToLittleOnGPU << <gdim, bdim >> > (d_cost, d_pki, start, n);
		CHECK(hipMemcpy(pki, d_pki, n_fl, hipMemcpyDeviceToHost));

		//for (int i = 0; i < 10; i++) { cout << pki[i] << " =? " << cost[i] <<endl; }

		//compute pk_mean
		integral(pki, pki_int, w, h);
		CHECK(hipMemcpy(d_pki_int, pki_int, n_fl, hipMemcpyHostToDevice));
		computeBoxFilterOnGPU << < gdim2, bdim2 >> > (d_pki, d_pki_int, d_pki_mean, (const int)w, (const int)h);

		//I*p
		pixelMultOnGPU << <gdim, bdim >> > (d_im, d_pki, d_convol, n);
		CHECK(hipMemcpy(convol, d_convol, n_fl, hipMemcpyDeviceToHost));

		//mean(I*p)
		integral(convol, convol_int, w, h);
		CHECK(hipMemcpy(d_convol_int, convol_int, n_fl, hipMemcpyHostToDevice));
		computeBoxFilterOnGPU << < gdim2, bdim2 >> > (d_convol, d_convol_int, d_convol_mean, (const int)w, (const int)h);

		//Compute ak and bk
		compute_ak_and_bk << <gdim, bdim >> > (d_mean_im, d_var_im, d_convol_mean, d_pki_mean, d_ak, d_bk, n);

		//compute ai, bi
		CHECK(hipMemcpy(ak, d_ak, n_fl, hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(bk, d_bk, n_fl, hipMemcpyDeviceToHost));
		integral(ak, ak_int, w, h);
		integral(bk, bk_int, w, h);
		CHECK(hipMemcpy(d_ak_int, ak_int, n_fl, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_bk_int, bk_int, n_fl, hipMemcpyHostToDevice));
		computeBoxFilterOnGPU << < gdim2, bdim2 >> > (d_ak, d_ak_int, d_ak_mean, (const int)w, (const int)h);
		computeBoxFilterOnGPU << < gdim2, bdim2 >> > (d_bk, d_bk_int, d_bk_mean, (const int)w, (const int)h);

		//compute qi
		compute_q << <gdim, bdim >> > (d_im, d_ak_mean, d_bk_mean, d_q, n);
		//int label = dmin + s;
		int label = dmin + s;
		//qi ->total filtered
		dispSelectOnGPU << <gdim, bdim >> > (d_q, d_filt_cost, d_dmap, (const int)n, label);
	}

	float* h_q = (float*)malloc(n_fl * sizeof(float));
	memset(h_q, 0, sizeof(float)*(n_fl));

	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(filter_cost, d_filt_cost, n_fl, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(disp_map, d_dmap, n_fl, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(h_q, d_q, n_fl, hipMemcpyDeviceToHost));

	for (size_t i = 0; i < n; i++)
	{
		cout << filter_cost[i] << endl;
	}

	//for (int s = 0; s < size_d; s++)
	//{
	//	int label = dmin + s;

	//	dispSelectOnCPU(h_q, h_filt_cost, disp_map, (const int)n, label);
	//}

	//

	//bool verif = check_errors(h_filt_cost, filter_cost, n_fl);
	//if (verif) cout << "Disp select ok!" << endl;

	//free(h_filt_cost);
	//free(h_q);

	//free cuda memory
	CHECK(hipFree(d_i));
	CHECK(hipFree(d_mean));
	CHECK(hipFree(d_im));
	CHECK(hipFree(d_mean_im));
	CHECK(hipFree(d_var_im));
	CHECK(hipFree(d_integral_im));
	CHECK(hipFree(d_integral_square));
	CHECK(hipFree(d_imSquare));
	CHECK(hipFree(d_temp));
	CHECK(hipFree(d_pki));
	CHECK(hipFree(d_ak));
	CHECK(hipFree(d_bk));
	CHECK(hipFree(d_ak_int));
	CHECK(hipFree(d_bk_int));
	CHECK(hipFree(d_ak_mean));
	CHECK(hipFree(d_bk_mean));
	CHECK(hipFree(d_pki_int));
	CHECK(hipFree(d_pki_mean));
	CHECK(hipFree(d_cost));
	CHECK(hipFree(d_filt_cost));
	CHECK(hipFree(d_dmap));
	CHECK(hipFree(d_convol_int));
	CHECK(hipFree(d_convol));
	CHECK(hipFree(d_convol_mean));
	CHECK(hipFree(d_q));

	//free ram memory
	free(h_im);
	free(h_mean_im);
	free(h_var_im);
	free(integral_im);
	free(integral_square);
	free(empty);
	free(big_empty);
	free(pki);
	free(pki_int);
	free(convol);
	free(convol_int);
	free(ak);
	free(ak_int);
	free(bk);
	free(bk_int);
}


__global__ void computeBoxFilterOnGPU(float* image, float* integral, float* mean, const int w, const int h) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int idy = blockIdx.y*blockDim.y + threadIdx.y;
	if (idx < w && idy < h) {
		mean[idx + w * idy] = computeMeanOnGPU(image, integral, idx, idy, w, h);
	}
}
__device__ float computeMeanOnGPU(float* I, float* S, int idx, int idy, const int w, const int h) {
	/**
	int i_x = max(idx - RADIUS, 0);
	int i_y = max(idy - RADIUS, 0);
	int j_x = min((idx + RADIUS), w-1);
	int j_y = min((idy + RADIUS), h-1);
	float S_1 = S[j_y*w + j_x];
	float S_2 = (i_x < 1) ? 0 : S[j_y*w + (i_x - 1)];
	float S_3 = (i_y < 1) ? 0 : S[(i_y - 1)*w + j_x];
	float S_4 = ((i_x < 1) || (i_y < 1)) ? 0 : S[(i_y - 1)*w + (i_x - 1)];
	float area = abs(j_y - i_y)*abs(j_x - i_x);
	return (S_1 + S_4 - S_3 - S_2) / area;
	**/
	int ymin = max(-1, idy - RADIUS - 1);
	int ymax = min(h - 1, idy + RADIUS);
	int xmin = max(-1, idx - RADIUS - 1);
	int xmax = min(w - 1, idx + RADIUS);
	float val = S[ymax*w + xmax];
	if (xmin >= 0)
		val -= S[ymax*w + xmin];
	if (ymin >= 0)
		val -= S[ymin*w + xmax];
	if (xmin >= 0 && ymin >= 0)
		val += S[ymin*w + xmin];
	return (1.0f*val / ((xmax - xmin)*(ymax - ymin)));
}

void computeBoxFilterOnCPU(float* image, float* integral, float* mean, const int w, const int h)
{
	for (size_t i = 0; i < w; i++)
	{
		for (size_t j = 0; j < h; j++)
		{
			mean[i + w * j] = computeMeanOnCPU(image, integral, i, j, w, h);
		}
	}
}
float computeMeanOnCPU(float* I, float* S, int idx, int idy, const int w, const int h)
{
	int ymin = max(-1, idy - RADIUS - 1);
	int ymax = min(h - 1, idy + RADIUS);
	int xmin = max(-1, idx - RADIUS - 1);
	int xmax = min(w - 1, idx + RADIUS);
	float val = S[ymax*w + xmax];
	if (xmin >= 0)
		val -= S[ymax*w + xmin];
	if (ymin >= 0)
		val -= S[ymin*w + xmax];
	if (xmin >= 0 && ymin >= 0)
		val += S[ymin*w + xmin];
	return (1.0f*val / ((xmax - xmin)*(ymax - ymin)));
}

// q calculations

__global__ void compute_ak(float* mean, float* var, float* convol, float* pk, float* a, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float c;
	if (i < len)
	{
		c = 1.0f / (var[i] + EPS);
		a[i] = 1.0f*(convol[i] - mean[i] * pk[i]) / c;
	}
}

__global__ void compute_ak_and_bk(float* mean, float* var, float* convol, float* pk, float* a, float* b, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float c;
	if (i < len)
	{
		c = 1.0f / (var[i] + EPS);
		a[i] = 1.0f*(convol[i] - mean[i] * pk[i]) / c;
		b[i] = 1.0f*pk[i] - 1.0f*mean[i] * a[i];
	}
}

__global__ void compute_bk(float* mean, float* a, float* pk, float* b, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		b[i] = 1.0f*pk[i] - 1.0f*mean[i] * a[i];
	}
}
__global__ void compute_q(float* im, float* a, float* b, float* q, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		q[i] = a[i] * im[i] + b[i];
	}
}

// CPU
void compute_akOnCPU(float* mean, float* var, float* convol, float* pk, float* a, int len) {
	for (size_t i = 0; i < len; i++)
	{
		float c = 1.0f / (var[i] + EPS);
		a[i] = 1.0f*(convol[i] - mean[i] * pk[i]) / c;
	}
}

void compute_ak_and_bkOnCPU(float* mean, float* var, float* convol, float* pk, float* a, float* b, int len) {
	for (size_t i = 0; i < len; i++)
	{
		float c = 1.0f / (var[i] + EPS);
		a[i] = 1.0f*(convol[i] - mean[i] * pk[i]) / c;
		b[i] = 1.0f*pk[i] - 1.0f*mean[i] * a[i];
	}
}

void compute_bkOnCPU(float* mean, float* a, float* pk, float* b, int len) {
	for (size_t i = 0; i < len; i++)
	{
		b[i] = 1.0f*pk[i] - 1.0f*mean[i] * a[i];
	}
}
void compute_qOnCPU(float* im, float* a, float* b, float* q, int len) {
	for (size_t i = 0; i < len; i++)
	{
		q[i] = a[i] * im[i] + b[i];
	}
}

//disp selection
__global__ void dispSelectOnGPU(float* q, float* filter_cost, float* dmap, const int n, int label) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id < n) {
		if (1.0f*filter_cost[id] >= 1.0f*q[id]) {
			dmap[id] = label;
			filter_cost[id] = q[id];
		}
	}
}

void dispSelectOnCPU(float* q, float* filter_cost, float* dmap, const int n, int label)
{
	for (size_t i = 0; i < n; i++)
	{
		if (1.0f*filter_cost[i] >= 1.0f*q[i]) {
			dmap[i] = label;
			filter_cost[i] = q[i];
		}
	}
}


// simple operations"
// GPU
__global__ void copyFromBigToLittleOnGPU(float* image1, float* result, int start, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		result[i] = image1[start + i];
	}
}
__global__ void copyFromLittleToBigOnGPU(float* image1, float* result, int start, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		result[i + start] = image1[i];
	}
}

__global__ void chToFlOnGPU(unsigned char* image, float* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		unsigned int c = image[i];
		result[i] = 1.0f*c;
	}
}

__global__ void flToChOnGPU(float* image, unsigned char* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		int c = image[i];
		result[i] = (c > 255) ? 255 : (unsigned char)c;
	}
}

__global__ void pixelMultOnGPU(float* image1, float* image2, float* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		result[i] = image1[i] * image2[i];
	}
}

__global__ void pixelSousOnGPU(float* image1, float* image2, float* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		result[i] = image1[i] - image2[i];
	}
}

__global__ void pixelAddOnGPU(float* image1, float* image2, float* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		result[i] = image1[i] + image2[i];
	}
}

__global__ void pixelDivOnGPU(float* image1, float* image2, float* result, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
	{
		float c = image2[i];
		if (c != 0) result[i] = image1[i] / c;
	}
}

//CPU functions
__host__ void chToFlOnCPU(unsigned char* image, float* result, int len) {
	for (int i = 0; i < len; i++) {
		unsigned int c = image[i];
		result[i] = 1.0f*c;
	}
}

__host__ void flToChOnCPU(float* image, unsigned char* result, int len) {
	for (int i = 0; i < len; i++)
	{
		unsigned int c = image[i];
		result[i] = (unsigned char)c;
	}
}

__host__ void pixelMultOnCPU(float* image1, float* image2, float* result, int len) {
	for (int i = 0; i < len; i++)
	{
		result[i] = image1[i] * image2[i];
	}
}

__host__ void pixelSousOnCPU(float* image1, float* image2, float* result, int len) {
	for (int i = 0; i < len; i++)
	{
		result[i] = image1[i] - image2[i];
	}
}

__host__ void pixelAddOnCPU(float* image1, float* image2, float* result, int len) {
	for (int i = 0; i < len; i++)
	{
		result[i] = image1[i] + image2[i];
	}
}

__host__ void pixelDivOnCPU(float* image1, float* image2, float* result, int len) {
	for (int i = 0; i < len; i++)
	{
		float c = image2[i];
		if (c != 0) result[i] = image1[i] / c;
	}
}