#include "hip/hip_runtime.h"
#include "occlusion.cuh"

__global__ void detect_occlusionOnGPU(float* disparityLeft, float* disparityRight, const float dOcclusion, const int dLR, const int w, const int h)
{
	int tdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (tdx >= w * h) return;

	int d = (int)disparityLeft[tdx];
	int dprime = (int)disparityRight[tdx];
	if ((tdx % w) + d < 0 || (tdx % w) + d >= w || abs(d + dprime) > dLR)
		disparityLeft[tdx] = dOcclusion;
}

void detect_occlusion(float* disparityLeft, float* disparityRight, const float dOcclusion, const int dLR, const int w, const int h)
{
	float* d_disparityLeft;
	float* d_disparityRight;

	int n = w * h;

	memset(disparityLeft, 0, n * sizeof(float));
	CHECK(hipMalloc((void**)&d_disparityLeft, n * sizeof(float)));
	CHECK(hipMalloc((void**)&d_disparityRight, n * sizeof(float)));

	CHECK(hipMemcpy(d_disparityLeft, disparityLeft, n * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_disparityRight, disparityRight, n * sizeof(float), hipMemcpyHostToDevice));

	dim3 nThreadsPerBlock(1024);
	dim3 nBlocks((n + nThreadsPerBlock.x - 1) / nThreadsPerBlock.x);

	detect_occlusionOnGPU << <nBlocks, nThreadsPerBlock >> > (d_disparityLeft, d_disparityRight, dOcclusion, dLR, w, h);

	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(disparityLeft, d_disparityLeft, n * sizeof(float), hipMemcpyDeviceToHost));

	CHECK(hipFree(d_disparityLeft));
	CHECK(hipFree(d_disparityRight));

}

/// Detect left-right discrepancies in disparity and put incoherent pixels to
/// value \a dOcclusion in \a disparityLeft.
void detect_occlusionOnCPU(float* disparityLeft, float* disparityRight, const float dOcclusion, const int dLR, const int w, const int h)
{
	for (int y = 0; y < h; y++)
	{
		for (int x = 0; x < w; x++)
		{
			int d = (int)disparityLeft[x + w * y];
			int dprime = (int)disparityRight[x + w * y];
			if (x + d < 0 || x + d >= w || abs(d + dprime) > dLR)
				disparityLeft[x + w * y] = dOcclusion;
		}
	}
}