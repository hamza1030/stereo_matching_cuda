#include "hip/hip_runtime.h"
#include "costVolume.cuh"
__host__ int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

void compute_cost(unsigned char* i1, unsigned char* i2, float* cost, int w1, int w2, int h1, int h2, bool host_gpu_compare) {
	int size_d = D_MAX - D_MIN + 1;
	int size_cost = h1 * w1*size_d;
	unsigned char* d_i1;
	unsigned char* d_i2;
	float* h_cost = (float*)malloc(size_cost * sizeof(float));
	float* d_cost;
	memset(cost, 0, sizeof(float)*(size_cost));
	memset(h_cost, 0, sizeof(float)*(size_cost));

	CHECK(hipMalloc((unsigned char**)&d_i1, w1 * h1));
	CHECK(hipMalloc((unsigned char**)&d_i2, w2 * h2));
	CHECK(hipMalloc((void**)&d_cost, size_cost * sizeof(float)));
	CHECK(hipMemcpy(d_i1, i1, w1 * h1, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_i2, i2, w2 * h2, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_cost, cost, sizeof(float)*size_cost, hipMemcpyHostToDevice));
	dim3 blockDim(32, size_d);
	dim3 gridDim;
	//gridDim.x = (w1*h1 + blockDim.x - 1)/blockDim.x;
	gridDim.x = iDivUp(w1*h1, blockDim.x);
	gridDim.y = 1;//size_d;

	costVolumOnGPU2 << <gridDim, blockDim >> > (d_i1, d_i2, d_cost, w1, w2, h1, h2, size_d);
	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(cost, d_cost, size_cost * sizeof(float), hipMemcpyDeviceToHost));

	//host side
	if (host_gpu_compare) {
		costVolumeOnCPU(i1, i2, h_cost, w1, w2, h1, h2, size_d);
		bool verif = check_errors(h_cost, cost, size_cost);
		if (verif) cout << "Cost Volume ok!" << endl;
	}

	// free device global memory
	CHECK(hipFree(d_cost));
	CHECK(hipFree(d_i1));
	CHECK(hipFree(d_i2));
	free(h_cost);
}
void disparity_selection(float* filtered_cost, float* best_cost, float* disparity_map, const int w, const int h, bool host_gpu_compare) {
	const int size_d = D_MAX - D_MIN + 1;
	const int n = w * h;
	int n_fl = w * h * sizeof(float);
	float* d_filtered_cost;
	float* d_best_cost;
	float* d_dmap;
	CHECK(hipMalloc((void**)&d_best_cost, n_fl));
	CHECK(hipMalloc((void**)&d_filtered_cost, size_d*n_fl));
	CHECK(hipMalloc((void**)&d_dmap, n_fl));
	CHECK(hipMemcpy(d_filtered_cost, filtered_cost, size_d*n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_dmap, disparity_map, n_fl, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_best_cost, best_cost, n_fl, hipMemcpyHostToDevice));

	dim3 blockDim(1024);
	dim3 gridDim((n +blockDim.x -1)/blockDim.x);
	//gridDim.x = (w1*h1 + blockDim.x - 1)/blockDim.x;

	selectionOnGpu<< <gridDim, blockDim >> > (d_filtered_cost, d_best_cost, d_dmap, n, size_d);



	CHECK(hipDeviceSynchronize());

	// check kernel error
	CHECK(hipGetLastError());

	CHECK(hipMemcpy(best_cost, d_best_cost, n_fl, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(disparity_map, d_dmap, n_fl, hipMemcpyDeviceToHost));


	// free device global memory
	CHECK(hipFree(d_best_cost));
	CHECK(hipFree(d_dmap));
	CHECK(hipFree(d_filtered_cost));
}

__global__ void selectionOnGpu(float* filt_cost, float* best_cost, float* dmap, const int n, const int dsize) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int offset = n;
	if (i < n) {
		for (int j = 0; j < dsize; j++) {
			if (best_cost[i] > filt_cost[i + j * n]) {
				best_cost[i] = filt_cost[i + j * n];
				dmap[i] = D_MIN + j;
			}
		}
	}
	
}



void costVolumeOnCPU(unsigned char* i1, unsigned char* i2, float* cost, int w1, int w2, int h1, int h2, int size_d) {
	float alpha = 1.0f*ALPHA;
	float th_color = 1.0f*TH_color;
	float th_grad = 1.0f*TH_grad;
	for (int d = -D_MIN; d <= D_MAX; d++) {
		for (int y = 0; y < h1; y++) {
			for (int x = 0; x < w1; x++) {
				int index = y * w1 + x;
				int id = d * w1*h1 + index;
				float c = (1.0f - alpha) * th_color + alpha * (1.0f*th_grad);
				if ((x + d < w2) && (x + d >= 0)) {
					float diff_term = 1.0f*abs(i1[index] - i2[index + d]);
					float grad_1 = 1.0f*x_derivativeCPU(i1, x, index, w1);
					float grad_2 = 1.0f*x_derivativeCPU(i2, x + d, index + d, w2);
					float grad_term = 1.0f*abs(grad_1 - grad_2);
					c = (1 - alpha)*min(diff_term, th_color) + alpha * min(grad_term, 1.0f*th_grad);
				}
				cost[id] = c;
			}
		}
	}
}

__global__ void costVolumOnGPU2(unsigned char* i1, unsigned char* i2, float* cost, int w1, int w2, int h1, int h2, int size_d) {
	// x threads for pixels [0, w*h]
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	// y threads for d [0, size_d]
	int y = blockDim.y*blockIdx.y + threadIdx.y;

	float alpha = 1.0f*ALPHA;
	float th_color = 1.0f*TH_color;
	float th_grad = 1.0f*TH_grad;

	// row index in the image
	int idx = x % w1;
	// col index in the image
	int idy = x / w1;
	// index [0, w*h*size_d]
	int id = y * w1*h1 + x;
	// d candidate [dmin, dmax]
	int d = -D_MIN + y;

	if (y < size_d && x < w1*h1) {
		// threshold
		float c = (1 - alpha) * th_color + alpha * th_grad;
		if (((idx + d) < w2) && ((idx + d) >= 0)) 
		{
			c = (1 - alpha)*difference_term(i1[x], i2[x + d]) + alpha * difference_term_2(x_derivative(i1, idx, x, w1), x_derivative(i2, idx + d, x + d, w2));
		}
		cost[id] = c;
		//printf("%f\n", c);

		//float* q;
		//// TODO filter
		//q[id] = 0;

		//__syncthreads();

		//// disparity selection - blockDim should be SIZE_1D !!!

		//// fill with 0
		//__shared__ float bestDisparity[SIZE_1D];
		//// fill with 100000
		//__shared__ float bestCost[SIZE_1D];

		//bestDisparity[threadIdx.x] = 0;
		//bestCost[threadIdx.x] = 0;

		//__syncthreads();

		//if (q[id] < bestCost[threadIdx.x])
		//{
		//	bestCost[threadIdx.x] = q[id];
		//	bestDisparity[threadIdx.x] = d;
		//}

		//__syncthreads();

		//// output to add in param - size w*h - fill with 0
		//float* disparityMap;
		//disparityMap[x] = bestDisparity[threadIdx.x];
	}

	//extern __shared__ float temp[];
	//// for shared memory
	//int tdx = threadIdx.x;
	//// to cumSum one row - for w = 1080, we need 540 threads
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//// for each row
	//int idy = blockIdx.y * blockDim.y + threadIdx.y;

	//int idxEven = idx * 2 + idy * w;
	//int idxOdd = idx * 2 + 1 + idy * w;

	//int offset = 1;

	//temp[2 * tdx] = input[idxEven];
	//temp[2 * tdx + 1] = input[idxOdd];

	//for (int nSum = B_SIZE / 2; nSum > 0; nSum /= 2)
	//{ 
	//	__syncthreads();
	//	if (tdx < nSum)
	//	{
	//		int a = offset * (2 * tdx + 1) - 1;
	//		int b = offset * (2 * tdx + 2) - 1;
	//		temp[b] += temp[a];
	//	}
	//	offset *= 2;
	//}

	//__syncthreads();

	////Write output (size h)
	//output[2 * tdx] = temp[2 * tdx];
	//output[2 * tdx + 1] = temp[2 * tdx + 1];
}

__device__ int id_im(int i, int j, int width) {
	return j * width + i;
}
__device__ int id_cost(int i, int j, int width, int height, int k) {
	return k * width*height + j * width + i;
}
__device__ float x_derivative(unsigned char* im, int col_index, int index, int width) {
	if ((col_index + 1) < width && (col_index - 1) >= 0)
	{
		return (float)((im[index + 1] - im[index - 1]) / 2);
	}
	else if (col_index + 1 == width)
	{
		return (float)((im[index] - im[index - 1]) / 2);
	}
	else
	{
		return (float)((im[index + 1] - im[index]) / 2);
	}
}

__host__ float x_derivativeCPU(unsigned char* im, int col_index, int index, int width) {
	if ((col_index + 1) < width && (col_index - 1) >= 0)
	{
		return (float)((im[index + 1] - im[index - 1]) / 2);
	}
	else if (col_index + 1 == width)
	{
		return (float)((im[index] - im[index - 1]) / 2);
	}
	else
	{
		return (float)((im[index + 1] - im[index]) / 2);
	}
}

__device__ int difference_term(unsigned char pixel_i, unsigned char pixel_j) {
	return min(abs((int)(pixel_i - pixel_j)), TH_color);
}
__device__ float difference_term_2(float pixel_i, float pixel_j) {
	return min(abs(pixel_i - pixel_j), 1.0f*TH_grad);
}

__device__ int getGlobalIdx_1D_2D()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}