#include "hip/hip_runtime.h"
#include "filter.cuh"

/**
__host__ void boxFilterOnCpu(char*in, char* out, int radius)
{
	double* S = new double[w*h]; // Use double to mitigate precision loss
	for (int i = w * h - 1; i >= 0; i--)
		S[i] = static_cast<double>(tab[i]);

	//cumulative sum table S, eq. (24)
	for (int y = 0; y<h; y++) { //horizontal
		double *in = S + y * w, *out = in + 1;
		for (int x = 1; x<w; x++)
			*out++ += *in++;
	}
	for (int y = 1; y<h; y++) { //vertical
		double *in = S + (y - 1)*w, *out = in + w;
		for (int x = 0; x<w; x++)
			*out++ += *in++;
	}

	//box filtered image B
	Image B(w, h);
	float *out = B.tab;
	for (int y = 0; y<h; y++) {
		int ymin = std::max(-1, y - radius - 1);
		int ymax = std::min(h - 1, y + radius);
		for (int x = 0; x<w; x++, out++) {
			int xmin = std::max(-1, x - radius - 1);
			int xmax = std::min(w - 1, x + radius);
			// S(xmax,ymax)-S(xmin,ymax)-S(xmax,ymin)+S(xmin,ymin), eq. (25)
			double val = S[ymax*w + xmax];
			if (xmin >= 0)
				val -= S[ymax*w + xmin];
			if (ymin >= 0)
				val -= S[ymin*w + xmax];
			if (xmin >= 0 && ymin >= 0)
				val += S[ymin*w + xmin];
			*out = static_cast<float>(val / ((xmax - xmin)*(ymax - ymin))); //average
		}
	}
	delete[] S;
	return B;
}
**/
/**

__host__ void covarianceOnCpu(char* I, char* out, int radius, char* mean) {
	var = boxFilter(I*I, char* out, radius);

	return boxFilter(r) - mean1 * mean2;
}
**/

__global__ void boxFilterOnGPU(unsigned char* image, unsigned char* mean, int width, int height) {
	int i = blockIdx.x * TILE_WIDTH + threadIdx.x - RADIUS;
	int j = blockIdx.y * TILE_HEIGHT + threadIdx.y - RADIUS;
	int ind = j * width + i;
	__shared__ float sharedMem[B_W][B_H];
	if (i < 0 || j < 0 || i >= width || j >= height) {
		sharedMem[threadIdx.x][threadIdx.y] = 0;
		return;
	}
	sharedMem[threadIdx.x][threadIdx.y] = image[ind];
	

	__syncthreads();

	// box filter (only for threads inside the tile)
	if ((threadIdx.x >= RADIUS) && (threadIdx.x < (B_W - RADIUS)) && (threadIdx.y >= RADIUS) && (threadIdx.y < (B_H - RADIUS))) {
		float sum = 0;
		for (int ix = -RADIUS; ix <= RADIUS; ix++) {
			for (int iy = -RADIUS; iy <= RADIUS; iy++) {
				sum += sharedMem[threadIdx.x + ix][threadIdx.y + iy];
			}
		}
		int val = sum / ((2 * RADIUS + 1)*(2 * RADIUS + 1));
		mean[ind] = (unsigned char)val;
		//mean[ind] = 220;
	}
}
__global__ void boxFilterfloatOnGpu(float* image, float* mean, int width, int height) {
	int i = blockIdx.x * TILE_WIDTH + threadIdx.x - RADIUS;
	int j = blockIdx.y * TILE_HEIGHT + threadIdx.y - RADIUS;
	int ind = j * width + i;

	__shared__ float sharedMem[B_W][B_H];
	if (i < 0 || j < 0 || i >= width || j >= height) { 
		sharedMem[threadIdx.x][threadIdx.y] = 0;
		return;
	}

	sharedMem[threadIdx.x][threadIdx.y] = image[ind];

	__syncthreads();

	// box filter (only for threads inside the tile)
	if ((threadIdx.x >= RADIUS) && (threadIdx.x < (B_W - RADIUS)) && (threadIdx.y >= RADIUS) && (threadIdx.y < (B_H - RADIUS))) {
		float sum = 0;
		for (int ix = -RADIUS; ix <= RADIUS; ix++) {
			for (int iy = -RADIUS; iy <= RADIUS; iy++) {
				sum += sharedMem[threadIdx.x + ix][threadIdx.y + iy];
			}
		}
		int val = sum / ((2 * RADIUS + 1)*(2 * RADIUS + 1));
		mean[ind] = val;
		//mean[ind] = 220;
	}
}

__global__ void multIm(unsigned char* im1, unsigned char* im2, float* val, int width, int height) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//N = (4180*2160)
	const int i = index;
	int N = width * height;
	if (i < N)
	{
		val[i] = im1[index] * im2[index];
	}
}

__global__ void sousIm(float* im1, float* im2, float* val, int width, int height) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//N = (4180*2160)
	int i = index;
	int N = width * height;
	if (i < N)
	{
		val[i] = im1[index] - im2[index];
	}
}

void filter(unsigned char* image, int width, int height, unsigned char* mean, float* var, bool cuda)
{
	const int size = 2 * RADIUS + 1;

	int n = width * height;
	memset(mean, 0, n);
	memset(var, 0, sizeof(float)*n);


	unsigned char* d_image;
	unsigned char* d_mean;
	float* d_mean2;
	float* d_var;
	float* d_mult_mean;
	float* d_mult_im;

	cout << "..........." << endl;

	// malloc device global memory
	CHECK(hipMalloc((unsigned char**)&d_image, n));
	CHECK(hipMalloc((unsigned char**)&d_mean, n));
	CHECK(hipMalloc((void**)&d_var, n * sizeof(float)));
	CHECK(hipMalloc((void**)&d_mult_mean, n * sizeof(float)));
	CHECK(hipMalloc((void**)&d_mult_im, n * sizeof(float)));
	CHECK(hipMalloc((void**)&d_mean2, n * sizeof(float)));

	CHECK(hipMemcpy(d_image, image, n, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_mean, mean, n, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_var, var, sizeof(float)*n, hipMemcpyHostToDevice));

	dim3 blockDim(B_W, B_H);
	int grid_w = width / TILE_WIDTH + 1;
	int grid_h = height / TILE_HEIGHT + 1;

	dim3 gridDim(grid_w, grid_h);
	boxFilterOnGPU << <gridDim, blockDim >> > (d_image, d_mean, width, height);


	//if (host_gpu_compare) {
	unsigned char* h_mean = (unsigned char*)malloc(n * sizeof(unsigned char));
	memset(h_mean, 0, sizeof(unsigned char)*(n));

	boxFilterOnCPU(image, h_mean, width, height);
	bool verif = check_errors(h_mean, mean, n);
	if (verif) cout << "Cost Volume ok!" << endl;

	free(h_mean);
	//}


	blockDim.x =1024;
	blockDim.y = 1;

	gridDim.x = ((n + blockDim.x - 1) / blockDim.x);
	gridDim.y = 1;
	multIm << <gridDim, blockDim >> > (d_image, d_image, d_mult_im, width, height);
	multIm << <gridDim, blockDim >> > (d_mean, d_mean, d_mult_mean, width, height);

	blockDim.x = B_W;
	blockDim.y = B_H;

	gridDim.x = grid_w;
	gridDim.y = grid_h;
	boxFilterfloatOnGpu << <gridDim, blockDim >> > (d_mult_im, d_mean2, width, height);

	blockDim.x = 1024;
	blockDim.y = 1;
	gridDim.x = ((n + blockDim.x - 1) / blockDim.x);
	gridDim.y = 1;
	sousIm << <gridDim, blockDim >> > (d_mean2, d_mult_mean, d_var, width, height);

	CHECK(hipDeviceSynchronize());
	//covarOnGpu << <gridDim, blockDim >> > (d_image, d_mean, d_var, width, height);

	// check kernel error
	CHECK(hipGetLastError());

	// copy kernel result back to host side
	CHECK(hipMemcpy(mean, d_mean, n, hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(var, d_var, sizeof(float)*n, hipMemcpyDeviceToHost));

	// free device global memory
	CHECK(hipFree(d_image));
	CHECK(hipFree(d_var));
	CHECK(hipFree(d_mean));
	CHECK(hipFree(d_mult_im));
	CHECK(hipFree(d_mean2));
	CHECK(hipFree(d_mult_mean));

	
}